#include "hip/hip_runtime.h"
/**
 *  Copyright (c) 2023 by Contributors
 *  Copyright (c) 2023, GT-TDAlab (Muhammed Fatih Balin & Umit V. Catalyurek)
 * @file cuda/index_select_csc_impl.cu
 * @brief Index select csc operator implementation on CUDA.
 */
#include <c10/core/ScalarType.h>
#include <c10/cuda/CUDAStream.h>
#include <graphbolt/cuda_ops.h>
#include <thrust/execution_policy.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>

#include <hipcub/hipcub.hpp>
#include <numeric>

#include "./common.h"
#include "./utils.h"

namespace graphbolt {
namespace ops {

constexpr int BLOCK_SIZE = 128;

// Given the in_degree array and a permutation, returns in_degree of the output
// and the permuted and modified in_degree of the input. The modified in_degree
// is modified so that there is slack to be able to align as needed.
template <typename indptr_t, typename indices_t>
struct AlignmentFunc {
  static_assert(GPU_CACHE_LINE_SIZE % sizeof(indices_t) == 0);
  const indptr_t* in_degree;
  const int64_t* perm;
  int64_t num_nodes;
  __host__ __device__ auto operator()(int64_t row) {
    constexpr int num_elements = GPU_CACHE_LINE_SIZE / sizeof(indices_t);
    return thrust::make_tuple(
        in_degree[row],
        // A single cache line has num_elements items, we add num_elements - 1
        // to ensure there is enough slack to move forward or backward by
        // num_elements - 1 items if the performed access is not aligned.
        (indptr_t)(in_degree[perm ? perm[row % num_nodes] : row] + num_elements - 1));
  }
};

template <typename indptr_t, typename indices_t>
__global__ void _CopyIndicesAlignedKernel(
    const indptr_t edge_count, const int64_t num_nodes,
    const indptr_t* const indptr, const indptr_t* const output_indptr,
    const indptr_t* const output_indptr_aligned, const indices_t* const indices,
    indices_t* const output_indices, const int64_t* const perm) {
  indptr_t idx = static_cast<indptr_t>(blockIdx.x) * blockDim.x + threadIdx.x;
  const int stride_x = gridDim.x * blockDim.x;

  while (idx < edge_count) {
    const auto permuted_row_pos =
        cuda::UpperBound(output_indptr_aligned, num_nodes, idx) - 1;
    const auto row_pos = perm ? perm[permuted_row_pos] : permuted_row_pos;
    const auto out_row = output_indptr[row_pos];
    const auto d = output_indptr[row_pos + 1] - out_row;
    const int offset =
        ((size_t)(indices + indptr[row_pos] - output_indptr_aligned[permuted_row_pos]) %
         GPU_CACHE_LINE_SIZE) /
        sizeof(indices_t);
    const auto rofs = idx - output_indptr_aligned[permuted_row_pos] - offset;
    if (rofs >= 0 && rofs < d) {
      const auto in_idx = indptr[row_pos] + rofs;
      assert((size_t)(indices + in_idx - idx) % GPU_CACHE_LINE_SIZE == 0);
      const auto u = indices[in_idx];
      output_indices[out_row + rofs] = u;
    }
    idx += stride_x;
  }
}

// Given rows and indptr, computes:
// inrow_indptr[i] = indptr[rows[i]];
// in_degree[i] = indptr[rows[i] + 1] - indptr[rows[i]];
template <typename indptr_t, typename nodes_t>
struct SliceFunc {
  const nodes_t* rows;
  const indptr_t* indptr;
  indptr_t* in_degree;
  indptr_t* inrow_indptr;
  __host__ __device__ auto operator()(int64_t tIdx) {
    const auto out_row = rows[tIdx];
    const auto indptr_val = indptr[out_row];
    const auto degree = indptr[out_row + 1] - indptr_val;
    in_degree[tIdx] = degree;
    inrow_indptr[tIdx] = indptr_val;
  }
};

struct PairSum {
  template <typename indptr_t>
  __host__ __device__ auto operator()(
      const thrust::tuple<indptr_t, indptr_t> a,
      const thrust::tuple<indptr_t, indptr_t> b) {
    return thrust::make_tuple(
        thrust::get<0>(a) + thrust::get<0>(b),
        thrust::get<1>(a) + thrust::get<1>(b));
  };
};

// Returns (indptr[nodes + 1] - indptr[nodes], indptr[nodes])
std::tuple<torch::Tensor, torch::Tensor> SliceCSCIndptr(
    torch::Tensor indptr, torch::Tensor nodes) {
  auto allocator = cuda::GetAllocator();
  const auto exec_policy =
      thrust::cuda::par_nosync(allocator).on(cuda::GetCurrentStream());
  const int64_t num_nodes = nodes.size(0);
  // Read indptr only once in case it is pinned and access is slow.
  auto sliced_indptr =
      torch::empty(num_nodes, nodes.options().dtype(indptr.scalar_type()));
  // compute in-degrees
  auto in_degree =
      torch::empty(num_nodes + 1, nodes.options().dtype(indptr.scalar_type()));
  thrust::counting_iterator<int64_t> iota(0);
  AT_DISPATCH_INTEGRAL_TYPES(
      indptr.scalar_type(), "IndexSelectCSCIndptr", ([&] {
        using indptr_t = scalar_t;
        AT_DISPATCH_INDEX_TYPES(
            nodes.scalar_type(), "IndexSelectCSCNodes", ([&] {
              using nodes_t = index_t;
              thrust::for_each(
                  exec_policy, iota, iota + num_nodes,
                  SliceFunc<indptr_t, nodes_t>{
                      nodes.data_ptr<nodes_t>(), indptr.data_ptr<indptr_t>(),
                      in_degree.data_ptr<indptr_t>(),
                      sliced_indptr.data_ptr<indptr_t>()});
            }));
      }));
  return {in_degree, sliced_indptr};
}

template <typename indptr_t, typename indices_t>
std::tuple<torch::Tensor, torch::Tensor> UVAIndexSelectCSCCopyIndices(
    torch::Tensor indices, const int64_t num_nodes,
    const indptr_t* const in_degree, const indptr_t* const sliced_indptr,
    const int64_t* const perm, torch::TensorOptions nodes_options,
    torch::ScalarType indptr_scalar_type, hipStream_t stream) {
  auto allocator = cuda::GetAllocator();
  thrust::counting_iterator<int64_t> iota(0);

  // Output indptr for the slice indexed by nodes.
  auto output_indptr =
      torch::empty(num_nodes + 1, nodes_options.dtype(indptr_scalar_type));

  auto output_indptr_aligned =
      allocator.AllocateStorage<indptr_t>(num_nodes + 1);

  {
    // Returns the actual and modified_indegree as a pair, the
    // latter overestimates the actual indegree for alignment
    // purposes.
    auto modified_in_degree = thrust::make_transform_iterator(
        iota, AlignmentFunc<indptr_t, indices_t>{in_degree, perm, num_nodes});
    auto output_indptr_pair = thrust::make_zip_iterator(
        output_indptr.data_ptr<indptr_t>(), output_indptr_aligned.get());
    thrust::tuple<indptr_t, indptr_t> zero_value{};
    // Compute the prefix sum over actual and modified indegrees.
    size_t tmp_storage_size = 0;
    CUDA_CALL(hipcub::DeviceScan::ExclusiveScan(
        nullptr, tmp_storage_size, modified_in_degree, output_indptr_pair,
        PairSum{}, zero_value, num_nodes + 1, stream));
    auto tmp_storage = allocator.AllocateStorage<char>(tmp_storage_size);
    CUDA_CALL(hipcub::DeviceScan::ExclusiveScan(
        tmp_storage.get(), tmp_storage_size, modified_in_degree,
        output_indptr_pair, PairSum{}, zero_value, num_nodes + 1, stream));
  }

  // Copy the actual total number of edges.
  auto edge_count =
      cuda::CopyScalar{output_indptr.data_ptr<indptr_t>() + num_nodes};
  // Copy the modified number of edges.
  auto edge_count_aligned =
      cuda::CopyScalar{output_indptr_aligned.get() + num_nodes};

  // Allocate output array with actual number of edges.
  torch::Tensor output_indices = torch::empty(
      static_cast<indptr_t>(edge_count),
      nodes_options.dtype(indices.scalar_type()));
  const dim3 block(BLOCK_SIZE);
  const dim3 grid(
      (static_cast<indptr_t>(edge_count_aligned) + BLOCK_SIZE - 1) /
      BLOCK_SIZE);

  // Perform the actual copying, of the indices array into
  // output_indices in an aligned manner.
  CUDA_KERNEL_CALL(
      _CopyIndicesAlignedKernel, grid, block, 0, stream,
      static_cast<indptr_t>(edge_count_aligned), num_nodes, sliced_indptr,
      output_indptr.data_ptr<indptr_t>(), output_indptr_aligned.get(),
      reinterpret_cast<indices_t*>(indices.data_ptr()),
      reinterpret_cast<indices_t*>(output_indices.data_ptr()), perm);
  return {output_indptr, output_indices};
}

std::tuple<torch::Tensor, torch::Tensor> UVAIndexSelectCSCImpl(
    torch::Tensor indptr, torch::Tensor indices, torch::Tensor nodes) {
  // Sorting nodes so that accesses over PCI-e are more regular.
  const auto sorted_idx =
      Sort(nodes, cuda::NumberOfBits(indptr.size(0) - 1)).second;
  auto stream = cuda::GetCurrentStream();
  const int64_t num_nodes = nodes.size(0);

  auto in_degree_and_sliced_indptr = SliceCSCIndptr(indptr, nodes);
  return AT_DISPATCH_INTEGRAL_TYPES(
      indptr.scalar_type(), "UVAIndexSelectCSCIndptr", ([&] {
        using indptr_t = scalar_t;
        auto in_degree =
            std::get<0>(in_degree_and_sliced_indptr).data_ptr<indptr_t>();
        auto sliced_indptr =
            std::get<1>(in_degree_and_sliced_indptr).data_ptr<indptr_t>();
        return GRAPHBOLT_DISPATCH_ELEMENT_SIZES(
            indices.element_size(), "UVAIndexSelectCSCCopyIndices", ([&] {
              return UVAIndexSelectCSCCopyIndices<indptr_t, element_size_t>(
                  indices, num_nodes, in_degree, sliced_indptr,
                  sorted_idx.data_ptr<int64_t>(), nodes.options(),
                  indptr.scalar_type(), stream);
            }));
      }));
}

template <typename indptr_t, typename indices_t>
struct IteratorFunc {
  indptr_t* indptr;
  indices_t* indices;
  __host__ __device__ auto operator()(int64_t i) { return indices + indptr[i]; }
};

template <typename indptr_t, typename indices_t>
struct ConvertToBytes {
  const indptr_t* in_degree;
  __host__ __device__ indptr_t operator()(int64_t i) {
    return in_degree[i] * sizeof(indices_t);
  }
};

template <typename indptr_t, typename indices_t>
void IndexSelectCSCCopyIndices(
    const int64_t num_nodes, indices_t* const indices,
    indptr_t* const sliced_indptr, const indptr_t* const in_degree,
    indptr_t* const output_indptr, indices_t* const output_indices,
    hipStream_t stream) {
  auto allocator = cuda::GetAllocator();
  thrust::counting_iterator<int64_t> iota(0);

  auto input_buffer_it = thrust::make_transform_iterator(
      iota, IteratorFunc<indptr_t, indices_t>{sliced_indptr, indices});
  auto output_buffer_it = thrust::make_transform_iterator(
      iota, IteratorFunc<indptr_t, indices_t>{output_indptr, output_indices});
  auto buffer_sizes = thrust::make_transform_iterator(
      iota, ConvertToBytes<indptr_t, indices_t>{in_degree});
  constexpr int64_t max_copy_at_once = std::numeric_limits<int32_t>::max();

  // Performs the copy from indices into output_indices.
  for (int64_t i = 0; i < num_nodes; i += max_copy_at_once) {
    size_t tmp_storage_size = 0;
    CUDA_CALL(hipcub::DeviceMemcpy::Batched(
        nullptr, tmp_storage_size, input_buffer_it + i, output_buffer_it + i,
        buffer_sizes + i, std::min(num_nodes - i, max_copy_at_once), stream));
    auto tmp_storage = allocator.AllocateStorage<char>(tmp_storage_size);
    CUDA_CALL(hipcub::DeviceMemcpy::Batched(
        tmp_storage.get(), tmp_storage_size, input_buffer_it + i,
        output_buffer_it + i, buffer_sizes + i,
        std::min(num_nodes - i, max_copy_at_once), stream));
  }
}

std::tuple<torch::Tensor, torch::Tensor> DeviceIndexSelectCSCImpl(
    torch::Tensor indptr, torch::Tensor indices, torch::Tensor nodes) {
  auto stream = cuda::GetCurrentStream();
  const int64_t num_nodes = nodes.size(0);
  auto in_degree_and_sliced_indptr = SliceCSCIndptr(indptr, nodes);
  return AT_DISPATCH_INTEGRAL_TYPES(
      indptr.scalar_type(), "IndexSelectCSCIndptr", ([&] {
        using indptr_t = scalar_t;
        auto in_degree =
            std::get<0>(in_degree_and_sliced_indptr).data_ptr<indptr_t>();
        auto sliced_indptr =
            std::get<1>(in_degree_and_sliced_indptr).data_ptr<indptr_t>();
        // Output indptr for the slice indexed by nodes.
        torch::Tensor output_indptr = torch::empty(
            num_nodes + 1, nodes.options().dtype(indptr.scalar_type()));

        {  // Compute the output indptr, output_indptr.
          size_t tmp_storage_size = 0;
          CUDA_CALL(hipcub::DeviceScan::ExclusiveSum(
              nullptr, tmp_storage_size, in_degree,
              output_indptr.data_ptr<indptr_t>(), num_nodes + 1, stream));
          auto allocator = cuda::GetAllocator();
          auto tmp_storage = allocator.AllocateStorage<char>(tmp_storage_size);
          CUDA_CALL(hipcub::DeviceScan::ExclusiveSum(
              tmp_storage.get(), tmp_storage_size, in_degree,
              output_indptr.data_ptr<indptr_t>(), num_nodes + 1, stream));
        }

        // Number of edges being copied.
        auto edge_count =
            cuda::CopyScalar{output_indptr.data_ptr<indptr_t>() + num_nodes};
        // Allocate output array of size number of copied edges.
        torch::Tensor output_indices = torch::empty(
            static_cast<indptr_t>(edge_count),
            nodes.options().dtype(indices.scalar_type()));
        GRAPHBOLT_DISPATCH_ELEMENT_SIZES(
            indices.element_size(), "IndexSelectCSCCopyIndices", ([&] {
              using indices_t = element_size_t;
              IndexSelectCSCCopyIndices<indptr_t, indices_t>(
                  num_nodes, reinterpret_cast<indices_t*>(indices.data_ptr()),
                  sliced_indptr, in_degree, output_indptr.data_ptr<indptr_t>(),
                  reinterpret_cast<indices_t*>(output_indices.data_ptr()),
                  stream);
            }));
        return std::make_tuple(output_indptr, output_indices);
      }));
}

std::tuple<torch::Tensor, torch::Tensor> IndexSelectCSCImpl(
    torch::Tensor indptr, torch::Tensor indices, torch::Tensor nodes) {
  if (indices.is_pinned()) {
    return UVAIndexSelectCSCImpl(indptr, indices, nodes);
  } else {
    return DeviceIndexSelectCSCImpl(indptr, indices, nodes);
  }
}

}  //  namespace ops
}  //  namespace graphbolt
