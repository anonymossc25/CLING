#include <c10/cuda/CUDACachingAllocator.h>
#include <hipcub/hipcub.hpp>
#include "atomic.h"
#include "cuda_common.h"
#include "tensor_ops.h"
#include "utils.h"

namespace gs {
namespace impl {

torch::Tensor SortIndicesCUDA(torch::Tensor indptr, torch::Tensor indices) {
  int64_t num_segments = indptr.numel() - 1;
  int64_t num_items = indices.numel();

  torch::Tensor sorted_indices = torch::empty_like(indices);

  void* d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;

  ID_TYPE_SWITCH(indptr.dtype(), IdType, {
    hipcub::DeviceSegmentedRadixSort::SortKeys(
        d_temp_storage, temp_storage_bytes, indices.data_ptr<IdType>(),
        sorted_indices.data_ptr<IdType>(), num_items, num_segments,
        indptr.data_ptr<IdType>(), indptr.data_ptr<IdType>() + 1);

    d_temp_storage =
        c10::cuda::CUDACachingAllocator::raw_alloc(temp_storage_bytes);

    hipcub::DeviceSegmentedRadixSort::SortKeys(
        d_temp_storage, temp_storage_bytes, indices.data_ptr<IdType>(),
        sorted_indices.data_ptr<IdType>(), num_items, num_segments,
        indptr.data_ptr<IdType>(), indptr.data_ptr<IdType>() + 1);
    c10::cuda::CUDACachingAllocator::raw_delete(d_temp_storage);
  });

  return sorted_indices;
}

}  // namespace impl
}  // namespace gs