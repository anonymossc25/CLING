
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello() {
    printf("Hello, CUDA!\n");
}

int main() {
    hello<<<1, 8>>>();
    hipDeviceSynchronize();
    return 0;
}

